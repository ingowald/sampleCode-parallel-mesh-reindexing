#include "hip/hip_runtime.h"
#include <owl/common/math/vec.h>
#include <owl/common/parallel/parallel_for.h>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>

#define CUDA_CHECK( call )                                              \
  {                                                                     \
    hipError_t rc = call;                                              \
    if (rc != hipSuccess) {                                            \
      fprintf(stderr,                                                   \
              "CUDA call (%s) failed with code %d (line %d): %s\n",     \
              #call, rc, __LINE__, hipGetErrorString(rc));             \
      OWL_RAISE("fatal cuda error");                                    \
    }                                                                   \
  }

#define CUDA_CALL(call) CUDA_CHECK(cuda##call)

#define CUDA_CHECK2( where, call )                                      \
  {                                                                     \
    hipError_t rc = call;                                              \
    if(rc != hipSuccess) {                                             \
      if (where)                                                        \
        fprintf(stderr, "at %s: CUDA call (%s) "                        \
                "failed with code %d (line %d): %s\n",                  \
                where,#call, rc, __LINE__, hipGetErrorString(rc));     \
      fprintf(stderr,                                                   \
              "CUDA call (%s) failed with code %d (line %d): %s\n",     \
              #call, rc, __LINE__, hipGetErrorString(rc));             \
      OWL_RAISE("fatal cuda error");                                    \
    }                                                                   \
  }

#define CUDA_SYNC_CHECK()                                       \
  {                                                             \
    hipDeviceSynchronize();                                    \
    hipError_t rc = hipGetLastError();                        \
    if (rc != hipSuccess) {                                    \
      fprintf(stderr, "error (%s: line %d): %s\n",              \
              __FILE__, __LINE__, hipGetErrorString(rc));      \
      OWL_RAISE("fatal cuda error");                            \
    }                                                           \
  }



using namespace owl::common;



__global__
void markUsed(int *isUsed, int *idx, int num)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid < num)
    isUsed[idx[tid]] = true;
}

__global__
void setNoDup(int *noDup, vec2f *vtx, int num)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid < num)
    noDup[tid]
      = (tid == 0)
      ? 1
      : (vtx[tid] != vtx[tid-1]);
}

__global__
void replaceUnused(vec2f *out_vtx,
                   int   *isUsed,
                   vec2f *in_vtx,
                   int   *in_idx,
                   int    numVertices)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numVertices) return;

  out_vtx[tid] = in_vtx[isUsed[tid]
                        ? tid
                        : (in_idx[0])];
}

__global__
void translateVertices(int *idx,
                       int *perm,
                       int *newIdx,
                       int numIndices)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numIndices) return;

  idx[tid] = newIdx[perm[idx[tid]]];
}

__global__
void setPerm(int *perm,
             int *orgID,
             int numVertices)
{
  int tid = threadIdx.x+blockIdx.x*blockDim.x;
  if (tid >= numVertices) return;

  perm[orgID[tid]] = tid;
}

template<typename T>
  struct subtract_one {
    inline __host__ __device__ T operator()(T &i) const { return i-(T)1; }
  };


template<typename T>
void print(const char *tag,
           const thrust::device_vector<T> &vec)
{
  std::cout << tag << "[" << vec.size() << "]\t: ";
  for (int i=0;i<vec.size();i++)
    std::cout << vec[i] << " ";
  std::cout << std::endl;
}

// remesh using cuda; modifying the arrays in place (but not shrinking
// the d_vtx array; vtxCount is input vertex count on the way in, and
// num actively used vertices on the way out; d_idx[] gets modified in
// place. Most d_vtx and d_idx must be device arrays
void remesh_cuda(thrust::device_vector<vec2f> &vtx,
                 thrust::device_vector<int>   &idx)
{
  thrust::device_vector<int> isUsed(vtx.size());
  thrust::fill(isUsed.begin(),isUsed.end(),0);

  // print("vtx in",vtx);
  // print("idx in",idx);
  
  markUsed<<<divRoundUp((int)idx.size(),1024),1024>>>
    (thrust::raw_pointer_cast(isUsed.data()),
     thrust::raw_pointer_cast(idx.data()),
     idx.size());
  //hipDeviceSynchronize();
  // print("used\t",isUsed);
  
  thrust::device_vector<vec2f> tmp_vtx(vtx.size());
  // print("pre-remunused",tmp_vtx);
  replaceUnused<<<divRoundUp((int)tmp_vtx.size(),1024),1024>>>
    (thrust::raw_pointer_cast(tmp_vtx.data()),
     thrust::raw_pointer_cast(isUsed.data()),
     thrust::raw_pointer_cast(vtx.data()),
     thrust::raw_pointer_cast(idx.data()),
     vtx.size());
  //hipDeviceSynchronize();
  // print("post-remunused",tmp_vtx);
  
  // ==================================================================
  // now, sort, and keep track of permutation done in sort
  // ==================================================================
  
  thrust::device_vector<int> orgID(vtx.size());
  thrust::sequence(orgID.begin(),orgID.end());

  // print("unsortedvtx",tmp_vtx);
  thrust::stable_sort_by_key(tmp_vtx.begin(),tmp_vtx.end(),orgID.data());
  // print("sorted vtx",tmp_vtx);

  // compute no dup array
  thrust::device_vector<int> noDup(vtx.size());
  setNoDup<<<divRoundUp((int)tmp_vtx.size(),1024),1024>>>
    (thrust::raw_pointer_cast(noDup.data()),
     thrust::raw_pointer_cast(tmp_vtx.data()),
     tmp_vtx.size());
  //hipDeviceSynchronize();
  // print("noDup",noDup);
  
  // postfix sum, and subtract one from each element
  thrust::device_vector<int> newIdx(vtx.size());
  thrust::inclusive_scan(noDup.begin(),noDup.end(),newIdx.data());
  thrust::transform(newIdx.begin(),newIdx.end(),newIdx.begin(),subtract_one<int>());
  hipDeviceSynchronize();

  // print("newIdx",newIdx);

  // get new num vertices
  int newN = newIdx.back()+1;
  // PRINT(newN);
  
  vtx.resize(newN);
  // ... and write new vertex array (we're writing back into vtx,
  // that's what we return to the app
  thrust::scatter(tmp_vtx.begin(),tmp_vtx.end(),newIdx.begin(),vtx.begin());
  // print("new vtx",vtx);
  // ==================================================================
  // new, clean vertex array created; this one contains neither
  // duplicates nor unused vertices. now about those indices...
  // ==================================================================
  
  thrust::device_vector<int> perm(orgID.size());
  setPerm<<<divRoundUp(int(orgID.size()),1024),1024>>>
    (thrust::raw_pointer_cast(perm.data()),
     thrust::raw_pointer_cast(orgID.data()),
     orgID.size());
  // print("orgID",orgID);
  // print("perm",perm);
  // print("newIdx",newIdx);
  
  // first, compute table to reverse the permutation
  translateVertices<<<divRoundUp(int(idx.size()),1024),1024>>>
    (thrust::raw_pointer_cast(idx.data()),
     thrust::raw_pointer_cast(perm.data()),
     thrust::raw_pointer_cast(newIdx.data()),
     idx.size());
                    
  // print("idx",idx);
}
